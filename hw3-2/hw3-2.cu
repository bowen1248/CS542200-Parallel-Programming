
#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <stdio.h>

#define BLOCK_SIZE 32
#define INF 1073741823 // 2^30 - 1

__device__ void block_APSP(int* C, int* A, int* B, int x, int y) {
    for (int k = 0; k < BLOCK_SIZE; k++) {
        // printf("%d %d %d %d %d %d\n", blockIdx.y, blockIdx.x, y, x, A[y * BLOCK_SIZE + k], B[k * BLOCK_SIZE + x]);
        C[y * BLOCK_SIZE + x] = min(C[y * BLOCK_SIZE + x], A[y * BLOCK_SIZE + k] + B[k * BLOCK_SIZE + x]);
        __syncthreads();
    }
}

__global__ void stage1(int *devMat, int startIdx, int n) {
    __shared__ int mat[BLOCK_SIZE * BLOCK_SIZE];

    // Load adj. matrix from global memory to shared memory
    int cursorX = startIdx + threadIdx.x;
    int cursorY = startIdx + threadIdx.y;
    mat[threadIdx.y * BLOCK_SIZE + threadIdx.x] = devMat[cursorY * n + cursorX];
    __syncthreads();

    // Perform APSP on the block
    block_APSP(mat, mat, mat, threadIdx.x, threadIdx.y);

    // Write data back to global memory
    devMat[cursorY * n + cursorX] = mat[threadIdx.y * BLOCK_SIZE + threadIdx.x];
    __syncthreads();
}

__global__ void stage2_row(int *devMat, int startIdx, int n) {
    // Load adj. matrix from global memory to shared memory
    int cursorX = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int cursorY = startIdx + threadIdx.y;

    if ((blockIdx.x * BLOCK_SIZE) == startIdx)
        return;

    // C[BLOCK_SIZE * BLOCK_SIZE], B[BLOCK_SIZE * BLOCK_SIZE]
    __shared__ int mat[2 * BLOCK_SIZE * BLOCK_SIZE];

    // Load adj. matrixs from global memory to shared memory
    mat[threadIdx.y * BLOCK_SIZE + threadIdx.x] = devMat[cursorY * n + cursorX];
    mat[threadIdx.y * BLOCK_SIZE + threadIdx.x + BLOCK_SIZE * BLOCK_SIZE] = devMat[cursorY * n + startIdx + threadIdx.x];
    __syncthreads();

    // Perform APSP on the block
    block_APSP(mat, &mat[BLOCK_SIZE * BLOCK_SIZE], mat, threadIdx.x, threadIdx.y);

    // Write data back to global memory
    devMat[cursorY * n + cursorX] = mat[threadIdx.y * BLOCK_SIZE + threadIdx.x];
    __syncthreads();
}

__global__ void stage2_col(int *devMat, int startIdx, int n) {
    // Load adj. matrix from global memory to shared memory
    int cursorX = startIdx + threadIdx.x;
    int cursorY = blockIdx.x * BLOCK_SIZE + threadIdx.y;
    
    if ((blockIdx.x * BLOCK_SIZE) == startIdx)
        return;

    // C[BLOCK_SIZE * BLOCK_SIZE], B[BLOCK_SIZE * BLOCK_SIZE]
    __shared__ int mat[2 * BLOCK_SIZE * BLOCK_SIZE];

    // Load adj. matrixs from global memory to shared memory
    mat[threadIdx.y * BLOCK_SIZE + threadIdx.x] = devMat[cursorY * n + cursorX];
    mat[threadIdx.y * BLOCK_SIZE + threadIdx.x + BLOCK_SIZE * BLOCK_SIZE] = devMat[(startIdx + threadIdx.y) * n + cursorX];
    __syncthreads();

    // Perform APSP on the block
    block_APSP(mat, mat, &mat[BLOCK_SIZE * BLOCK_SIZE], threadIdx.x, threadIdx.y);

    // Write data back to global memory
    devMat[cursorY * n + cursorX] = mat[threadIdx.y * BLOCK_SIZE + threadIdx.x];
    __syncthreads();
}

__global__ void stage3(int *devMat, int startIdx, int n) {
    // Load adj. matrix from global memory to shared memory
    int cursorX = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int cursorY = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    // long long int t1 = clock64();
    if ((blockIdx.x * BLOCK_SIZE) == startIdx || (blockIdx.y * BLOCK_SIZE) == startIdx)
        return;

    // C[BLOCK_SIZE * BLOCK_SIZE], B[BLOCK_SIZE * BLOCK_SIZE], A[BLOCK_SIZE * BLOCK_SIZE]
    __shared__ int mat[3 * BLOCK_SIZE * BLOCK_SIZE];

    // Load adj. matrixs from global memory to shared memory
    mat[threadIdx.y * BLOCK_SIZE + threadIdx.x] = devMat[cursorY * n + cursorX];
    mat[threadIdx.y * BLOCK_SIZE + threadIdx.x + BLOCK_SIZE * BLOCK_SIZE] = devMat[cursorY * n + startIdx + threadIdx.x];
    mat[threadIdx.y * BLOCK_SIZE + threadIdx.x + 2 * BLOCK_SIZE * BLOCK_SIZE] = devMat[(startIdx + threadIdx.y) * n + cursorX];
    __syncthreads();
    // long long int t2 = clock64();
    // printf("%lld ", (t2 - t1));
    // Perform APSP on the block
    block_APSP(mat, &mat[BLOCK_SIZE * BLOCK_SIZE], &mat[2 * BLOCK_SIZE * BLOCK_SIZE], threadIdx.x, threadIdx.y);
    // long long int t3 = clock64();
    // printf("%lld ", (t3 - t2));
    // Write data back to global memory
    devMat[cursorY * n + cursorX] = mat[threadIdx.y * BLOCK_SIZE + threadIdx.x];
    __syncthreads();
    // long long int t4 = clock64();
    // printf("%lld ", (t4 - t3));
}

int main(int argc, char **argv) {
    freopen("log.txt","w",stdout);
    /* detect how many CPUs are available */
    // cpu_set_t cpu_set;
    // int ncpus;
    // sched_getaffinity(0, sizeof(cpu_set), &cpu_set);
    // ncpus = CPU_COUNT(&cpu_set);

    // // Thread handlers
    // pthread_t threads[ncpus];

    /* argument parsing */
    assert(argc == 3);
    const char *inputFile = argv[1];
    const char *outputFile = argv[2];

    FILE *inFp = fopen(inputFile, "rb");
    FILE *outFp = fopen(outputFile, "wb");
    if( inFp == NULL ) {
        fprintf(stderr, "Couldn't open %s: %s\n", inputFile, strerror(errno));
        exit(1);
    }
    int verticesTotal;
    int edgesTotal;

    // Get input vertices and edges number
    size_t _;
    _ = fread(&verticesTotal, sizeof(int), 1, inFp);
    _ = fread(&edgesTotal, sizeof(int), 1, inFp);
    static int block_dim = (verticesTotal + BLOCK_SIZE - 1) / BLOCK_SIZE;
    static int n = BLOCK_SIZE * block_dim;

    // Create adjanency matrix
    int *adjMat = (int *) malloc(n * n * sizeof(int));

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i == j)
                adjMat[i * n + j] = 0;
            else
                adjMat[i * n + j] = INF;
        }
    }

    // Put edges into adjanency matrix
    int tmp[3];
    for (int i = 0; i < edgesTotal; i++) {
        _ = fread(&tmp, sizeof(int), 3, inFp);
        adjMat[tmp[0] * n + tmp[1]] = tmp[2];
    }
    fclose(inFp);

    // Print input graph
    // for (int i = 0; i < n; i++) {
    //     for (int j = 0; j < n; j++) {
    //         if (adjMat[i * n + j] != 1073741823)
    //             std::cout << adjMat[i * n + j] << " ";
    //         else
    //             std::cout << "INF" << " ";
    //     }
    //     std::cout << std::endl;
    // }

    // Stream number
    // const int nStreams = 8;

    // Init streams
    // cudaStream_t streams[nStreams];
    // for (int i = 0; i < nStreams; i++) {
    //     cudaStreamCreate(&streams[i]);
    // }

    int* device_adjMat;
    hipMalloc((void **) &device_adjMat, n * n * sizeof(int));

    // Put adjanency matrix to GPU
    hipMemcpy(device_adjMat, adjMat, n * n * sizeof(int), hipMemcpyHostToDevice);

    // stages
    for (int k_start = 0; k_start < n; k_start += BLOCK_SIZE) {
        stage1<<< 1, dim3(BLOCK_SIZE, BLOCK_SIZE), 0 >>> (device_adjMat, k_start, n);
        stage2_row<<< block_dim, dim3(BLOCK_SIZE, BLOCK_SIZE), 0 >>> (device_adjMat, k_start, n);
        stage2_col<<< block_dim, dim3(BLOCK_SIZE, BLOCK_SIZE), 0 >>> (device_adjMat, k_start, n);
        stage3<<< dim3(block_dim, block_dim), dim3(BLOCK_SIZE, BLOCK_SIZE), 0 >>> (device_adjMat, k_start, n);
    }

    // output
    hipMemcpy(adjMat, device_adjMat, n * n * sizeof(int), hipMemcpyDeviceToHost);

    // Print input graph
    // for (int i = 0; i < verticesTotal; i++) {
    //     for (int j = 0; j < verticesTotal; j++) {
    //         if (ansMat[i * n + j] != 1073741823)
    //             std::cout << ansMat[i * n + j] << " ";
    //         else
    //             std::cout << "INF" << " ";
    //     }
    //     std::cout << std::endl;
    // }

    for (int i = 0; i < verticesTotal; i++) {
        fwrite(&adjMat[i * n], sizeof(int), verticesTotal, outFp);
    }
    fclose(outFp);

    return 0;
}