
#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <stdio.h>

#define BLK_n 32
#define log2BLK_n 5
#define THREAD_NUMS 32
#define INF 1073741823 // 2^30 - 1

__global__ void stage1(int *devMat, int g_k, int n) {
    // Upperleft coordinate of this thread to global memory
    int g_x = g_k + threadIdx.x;
    int g_y = g_k + threadIdx.y;
    // Upperleft coordinate of this thread to shared memory
    int s_x = threadIdx.x;
    int s_y = threadIdx.y;

    __shared__ int mat[BLK_n * BLK_n];

    // Load adj. matrix from global memory to shared memory
    mat[s_y * BLK_n + s_x] = devMat[g_y * n + g_x];

    // Perform APSP on the block
    for (int k = 0; k < BLK_n; k++) {
        __syncthreads();
        mat[s_y * BLK_n + s_x] = min(mat[s_y * BLK_n + s_x], mat[s_y * BLK_n + k] + mat[k * BLK_n + s_x]);
    }

    // Write data back to global memory
    devMat[g_y * n + g_x] = mat[s_y * BLK_n + s_x];
}

__global__ void stage2(int *devMat, int g_k, int n) {
    // Matrix to be changed
    // Note blockidx.y 0 is row, 1 is column
    int s_x = threadIdx.x;
    int s_y = threadIdx.y;

    __shared__ int mat[2 * BLK_n * BLK_n];

    // Load adj. matrixs from global memory to shared memory
    if (blockIdx.y) {
        // column
        int g_x = g_k + s_x;
        int g_y = (blockIdx.x + (blockIdx.x >= (g_k >> log2BLK_n))) * BLK_n + s_y;

        mat[s_y * BLK_n + s_x] =  devMat[g_y * n + g_x];
        mat[s_y * BLK_n + s_x + BLK_n * BLK_n] = devMat[(g_k + s_y) * n + g_x];

        for (int k = 0; k < BLK_n; k++) {
            __syncthreads();
            mat[s_y * BLK_n + s_x] = min(mat[s_y * BLK_n + s_x], mat[s_y * BLK_n + k] + mat[k * BLK_n + s_x + BLK_n * BLK_n]);
        }

        // Write data back to global memory
        devMat[g_y * n + g_x] = mat[s_y * BLK_n + s_x];
    } else {
        // row
        int g_x = (blockIdx.x + (blockIdx.x >= (g_k >> log2BLK_n))) * BLK_n + s_x;
        int g_y = g_k + s_y;

        mat[s_y * BLK_n + s_x] = devMat[g_y * n + g_x];
        mat[s_y * BLK_n + s_x + BLK_n * BLK_n] = devMat[g_y * n + (g_k + s_x)];
        for (int k = 0; k < BLK_n; k++) {
            __syncthreads();
            mat[s_y * BLK_n + s_x] = min(mat[s_y * BLK_n + s_x], mat[s_y * BLK_n + k + BLK_n * BLK_n] + mat[k * BLK_n + s_x]);
        }

        // Write data back to global memory
        devMat[g_y * n + g_x] = mat[s_y * BLK_n + s_x];
    }
}

__global__ void stage3(int *devMat, int g_k, int n) {
    // Load adj. matrix from global memory to shared memory
    int s_x = threadIdx.x;
    int s_y = threadIdx.y;
    int g_x = (blockIdx.x + (blockIdx.x >= (g_k >> log2BLK_n))) * BLK_n + s_x;
    int g_y = (blockIdx.y + (blockIdx.y >= (g_k >> log2BLK_n))) * BLK_n + s_y;

    __shared__ int mat[2 * BLK_n * BLK_n];

    // Load adj. matrixs from global memory to shared memory
    int num = devMat[g_y * n + g_x];
    mat[s_y * BLK_n + s_x] = devMat[g_y * n + (g_k + s_x)];
    mat[s_y * BLK_n + s_x + BLK_n * BLK_n] = devMat[(g_k + s_y) * n + g_x];
    __syncthreads();

    // Perform APSP on the block
    for (int k = 0; k < BLK_n; k++) {
        num = min(num, mat[s_y * BLK_n + k] + mat[k * BLK_n + s_x + BLK_n * BLK_n]);
    }

    // Write data back to global memory
    devMat[g_y * n + g_x] = num;
    __syncthreads();
    // long long int t4 = clock64();
    // printf("%lld ", (t4 - t3));
}

int main(int argc, char **argv) {
    // freopen("log.txt","w",stdout);

    /* argument parsing */
    assert(argc == 3);
    const char *inputFile = argv[1];
    const char *outputFile = argv[2];

    FILE *inFp = fopen(inputFile, "rb");
    FILE *outFp = fopen(outputFile, "wb");

    int verticesTotal;
    int edgesTotal;
    
    // Get input vertices and edges number
    fread(&verticesTotal, sizeof(int), 1, inFp);
    fread(&edgesTotal, sizeof(int), 1, inFp);
    static int block_dim = (verticesTotal + BLK_n - 1) / BLK_n;
    static int n = BLK_n * block_dim;
    // printf("%d %d ", verticesTotal, edgesTotal);

    // Create adjanency matrix
    int *adjMat = (int *) malloc(n * n * sizeof(int));
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i == j)
                adjMat[i * n + j] = 0;
            else
                adjMat[i * n + j] = INF;
        }
    }

    // Put edges into adjanency matrix
    int tmp[15];
    int i; 
    for (i = 0; i < (edgesTotal - 5); i += 5) {
        fread(&tmp, sizeof(int), 15, inFp);
        adjMat[tmp[0] * n + tmp[1]] = tmp[2];
        adjMat[tmp[3] * n + tmp[4]] = tmp[5];
        adjMat[tmp[6] * n + tmp[7]] = tmp[8];
        adjMat[tmp[9] * n + tmp[10]] = tmp[11];
        adjMat[tmp[12] * n + tmp[13]] = tmp[14];
    }
    for (i = i; i < edgesTotal; i += 1) {
        fread(&tmp, sizeof(int), 3, inFp);
        adjMat[tmp[0] * n + tmp[1]] = tmp[2];
    }
    fclose(inFp);

    // Print input graph
    // for (int i = 0; i < n; i++) {
    //     for (int j = 0; j < n; j++) {
    //         if (adjMat[i * n + j] != INF)
    //             std::cout << adjMat[i * n + j] << " ";
    //         else
    //             std::cout << "INF" << " ";
    //     }
    //     std::cout << std::endl;
    // }

    int* devMat;
    hipMalloc((void **) &devMat, n * n * sizeof(int));

    // Put adjanency matrix to GPU
    hipMemcpy(devMat, adjMat, n * n * sizeof(int), hipMemcpyHostToDevice);

    // stages
    for (int g_k = 0; g_k < n; g_k += BLK_n) {
        stage1<<< 1, dim3(THREAD_NUMS, THREAD_NUMS), 0 >>> (devMat, g_k, n);
        stage2<<< dim3(block_dim - 1, 2), dim3(THREAD_NUMS, THREAD_NUMS), 0 >>> (devMat, g_k, n);
        stage3<<< dim3(block_dim - 1, block_dim - 1), dim3(THREAD_NUMS, THREAD_NUMS), 0 >>> (devMat, g_k, n);
    }

    // output
    hipMemcpy(adjMat, devMat, n * n * sizeof(int), hipMemcpyDeviceToHost);

    // Print input graph
    // for (int i = 0; i < verticesTotal; i++) {
    //     for (int j = 0; j < verticesTotal; j++) {
    //         if (adjMat[i * n + j] != 1073741823)
    //             std::cout << adjMat[i * n + j] << " ";
    //         else
    //             std::cout << "INF" << " ";
    //     }
    //     std::cout << std::endl;
    // }

    for (int i = 0; i < verticesTotal; i++) {
        fwrite(&adjMat[i * n], sizeof(int), verticesTotal, outFp);
    }
    fclose(outFp);

    return 0;
}